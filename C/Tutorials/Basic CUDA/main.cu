#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

// Kernel function
__global__ void vectorAdd(int * a, int * b, int * c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    return;
}

int main() {

    int a[] = {1,2,3, 4,5,6, 7,8,9, 10,11,12, 13, 14, 15, 16, 17, 18};
    int b[] = {4,5,6, 7,8,9, 10,11,12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22};

    int c[sizeof(a)/sizeof(int)] = {0};

    // CPU example
    /*
    for(int i = 0; i < sizeof(c)/sizeof(int); i++) {
        c[i] = a[i] + b[i];
    }

    return;
     */

    // GPU example
    // Create pointers into the GPU memory
    int * cudaA = 0;
    int * cudaB = 0;
    int * cudaC = 0;

    // Allocate memory on the GPU
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // Copy data from the CPU to the GPU
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    // Launch the kernel
    // vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(cudaA, cudaB, cudaC);
    vectorAdd<<<1, sizeof(a)/sizeof(int)>>>(cudaA, cudaB, cudaC);

    // Copy data from the GPU to the CPU
    cudeMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    return;
}
